#include "hip/hip_runtime.h"
#inculde<stdio.h>

__global__ void add(int* a, int* b, int* c) {
    
    int i = threadIdx.x + blockDim.x * blockIdx.x ;
    c[i] = a[i] + b[i];

}

__managed__ int a[256], b[256], c[256];

int main() {

    for (int i = 0; i < 256; i++) {
        vector_a[i] = i;

        vector_b[i] = 256 - i;
    }

    

    add<<<1, 256 >>>(vector_a, vector_b, vector_c);
    
    hipDeviceSynchronize();
    
    int reuslt_sum = 0;
    
    for (int i = 0; i < 256; i++) {
        result_sum += vector_c[i];
    }

    printf("Finally I did it.\n");
    printf("Sum of the vector is %d\n", result_sum);

    return 0;

}